
#include <hip/hip_runtime.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100

/* this GPU kernel function calculates a random number and stores it in the parameter */
__global__ void random(int* result) {
	/* CUDA's random number library uses curandState_t to keep track of the seed value
	we will store a random state for every thread  */
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(clock(), /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	/* curand works like rand - except that it takes a state as a parameter */
	*result = hiprand(&state) % MAX;
}

int main() {
	/* allocate an int on the GPU */
	int* gpu_x;
	hipMalloc((void**)&gpu_x, sizeof(int));

	/* invoke the GPU to initialize all of the random states */
	random <<<1, 1>>>(gpu_x);

	/* copy the random number back */
	int x;
	hipMemcpy(&x, gpu_x, sizeof(int), hipMemcpyDeviceToHost);

	printf("Random number = %d.\n", x);

	/* free the memory we allocated */
	hipFree(gpu_x);

	return 0;
}